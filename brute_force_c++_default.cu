#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void bruteForceKernel(const char* characters, int charactersLength, const char* password, int passwordLength, bool* found, char* result, long long startAttempt, long long endAttempt, unsigned long long* attemptCount) {
    long long index = startAttempt + blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= endAttempt) return;

    char guess[32]; // Maksimum şifre uzunluğu
    long long temp = index;

    // Kombinasyon oluştur
    for (int i = 0; i < passwordLength; i++) {
        guess[i] = characters[temp % charactersLength];
        temp /= charactersLength;
    }
    guess[passwordLength] = '\0';

    // Şifre eşleşmesini kontrol et
    bool match = true;
    for (int i = 0; i < passwordLength; i++) {
        if (guess[i] != password[i]) {
            match = false;
            break;
        }
    }

    // Eşleşme bulunduysa sonucu kaydet
    if (match) {
        *found = true;
        for (int i = 0; i < passwordLength; i++) {
            result[i] = guess[i];
        }
        result[passwordLength] = '\0';
    }

    // Deneme sayısını artır
    atomicAdd(attemptCount, 1);
}

string bruteForce(const string& password, unsigned long long& totalAttempts) {
    const string characters = "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789";
    const int charactersLength = characters.length();
    const int passwordLength = password.length();

    totalAttempts = 1;
    for (int i = 0; i < passwordLength; i++) {
        totalAttempts *= charactersLength;
    }

    bool* dev_found;
    char* dev_result;
    char* dev_characters;
    char* dev_password;
    unsigned long long* dev_attemptCount;

    // Bellek ayırma
    hipMalloc((void**)&dev_found, sizeof(bool));
    hipMalloc((void**)&dev_result, (passwordLength + 1) * sizeof(char));
    hipMalloc((void**)&dev_characters, charactersLength * sizeof(char));
    hipMalloc((void**)&dev_password, passwordLength * sizeof(char));
    hipMalloc((void**)&dev_attemptCount, sizeof(unsigned long long));

    // Veriyi kopyalama
    hipMemcpy(dev_characters, characters.c_str(), charactersLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_password, password.c_str(), passwordLength * sizeof(char), hipMemcpyHostToDevice);

    bool found = false;
    hipMemcpy(dev_found, &found, sizeof(bool), hipMemcpyHostToDevice);

    unsigned long long attemptCount = 0;
    hipMemcpy(dev_attemptCount, &attemptCount, sizeof(unsigned long long), hipMemcpyHostToDevice);

    // CUDA çekirdeğini çağırma
    int blockSize = 256;
    long long attemptsPerKernel = 1000000; // Her kernel çağrısında 1 milyon deneme
    for (long long startAttempt = 0; startAttempt < totalAttempts; startAttempt += attemptsPerKernel) {
        long long endAttempt = min(startAttempt + attemptsPerKernel, totalAttempts);
        long long numBlocks = (endAttempt - startAttempt + blockSize - 1) / blockSize;
        bruteForceKernel<<<numBlocks, blockSize>>>(dev_characters, charactersLength, dev_password, passwordLength, dev_found, dev_result, startAttempt, endAttempt, dev_attemptCount);
        hipMemcpy(&found, dev_found, sizeof(bool), hipMemcpyDeviceToHost);
        if (found) break;
    }

    // Sonucu geri kopyalama
    char result[32];
    hipMemcpy(result, dev_result, (passwordLength + 1) * sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(&attemptCount, dev_attemptCount, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Belleği serbest bırakma
    hipFree(dev_found);
    hipFree(dev_result);
    hipFree(dev_characters);
    hipFree(dev_password);
    hipFree(dev_attemptCount);

    if (found) {
        cout << "Password found: " << result << endl;
        cout << "Total attempts: " << attemptCount << endl;
        return result;
    } else {
        cout << "Password not found." << endl;
        cout << "Total attempts: " << attemptCount << endl;
        return "";
    }
}

int main() {
    string password;
    cout << "Please enter a password: ";
    cin >> password;

    auto start = chrono::high_resolution_clock::now();
    unsigned long long totalAttempts;
    bruteForce(password, totalAttempts);
    auto end = chrono::high_resolution_clock::now();

    chrono::duration<double> elapsed = end - start;
    cout << "Elapsed time: " << elapsed.count() << " seconds" << endl;

    // Prevent the console from closing immediately
    cout << "Press any key to exit...";
    cin.ignore(); // Clear the previous input
    cin.get();    // Wait for the user to press a key

    return 0;
}